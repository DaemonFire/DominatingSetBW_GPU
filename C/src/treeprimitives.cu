#include "hip/hip_runtime.h"
#include "../include/treeprimitives.h"
#include "../include/algorithms.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <errno.h>
#include <sys/stat.h>


int getallleaves(dectree t, int *list){
	int n=0;
	int *lleft;
	int *lright;
	int nleft=0;
	int nright=0;
	if (t.left!=NULL){
		nleft= getnumberofleaves (*(t.left));
		lleft=(int*)malloc(nleft*sizeof(int));
		getallleaves(*(t.left), lleft);
	}

	if (t.right!=NULL){
		nright=getnumberofleaves(*(t.right));
		lright=(int*)malloc(nright*sizeof(int));
		getallleaves(*(t.right),lright);
	}

	if ((t.left==NULL)&&(t.right==NULL)){
		list[0]=t.label;
		n+=1;
	}
	else {
		n=nleft+nright;
		for (int i=0; i<nleft;i++)
			list[i]=lleft[i];

		for (int i=0;i<nright;i++)
			list[nleft+i]=lright[i];
	}
	return n;
}


int getnumberofleaves (dectree t){
	int n=0;
	int nleft=0;
	int nright=0;

	if (t.left!=NULL)
		nleft=getnumberofleaves(*(t.left));
	if (t.right!=NULL)
		nright=getnumberofleaves(*(t.right));
	if ((t.left==NULL)&&(t.right==NULL))
		n+=1;
	else 
		n=nleft+nright;
	return n;
}

dectree *generateTree (pointset p, graph g, int verthor){
	dectree *t;
	t = (dectree*)malloc(sizeof(dectree));
	pointset p1;
	pointset p2;
	p1.size=0;
	p2.size=0;
	p1.members=(int*)malloc(p.size*sizeof(int));
	p2.members=(int*)malloc(p.size*sizeof(int));
	int vnext=0;
	if (p.size==0)
		return NULL;
	else if (p.size==1){
		(*t).label=p.members[0];
		(*t).left=NULL;
		(*t).right=NULL;
	}
	else if (p.size==2){
		(*t).label=-1;
		(*t).left=(dectree*)malloc(sizeof(dectree));
		(*t).right=(dectree*)malloc(sizeof(dectree));
		dectree *t1;
		dectree *t2;
		t1=(dectree*)malloc(sizeof(dectree));
		t2=(dectree*)malloc(sizeof(dectree));
		(*t).left=t1;
		(*t).right=t2;
		(*t1).label=p.members[0];
		(*t1).left=NULL;
		(*t1).right=NULL;
		(*t2).label=p.members[1];
		(*t2).left=NULL;
		(*t2).right=NULL;
	}
	else {
		if (verthor==0){
			int xmoy=0;
			for (int i=0;i<p.size;i++)
				xmoy+=g.pos[2*p.members[i]];
			xmoy=xmoy/p.size;
			
			for (int i=0;i<p.size;i++){
				if (g.pos[2*p.members[i]]<xmoy){
					p1.size++;
					p1.members[p1.size-1]=p.members[i];
				}
				else {
					p2.size++;
					p2.members[p2.size-1]=p.members[i];
				}
			}
			if (p1.size==0){
				p1.size++;
				p1.members[0]=p2.members[p2.size-1];
				p2.size--;
			}
			if (p2.size==0){
				p2.size++;
				p2.members[0]=p1.members[p1.size-1];
				p1.size--;
			}
			vnext=1;
		}

		if (verthor==1){
			int ymoy=0;
			for (int i=0;i<p.size;i++)
				ymoy+=g.pos[2*p.members[i]+1];
			ymoy=ymoy/p.size;
			
			for (int i=0;i<p.size;i++){
				if (g.pos[2*p.members[i]+1]<ymoy){
					p1.size++;
					p1.members[p1.size-1]=p.members[i];
				}
				else {
					p2.size++;
					p2.members[p2.size-1]=p.members[i];
				}
			}
			if (p1.size==0){
				p1.size++;
				p1.members[0]=p2.members[p2.size-1];
				p2.size--;
			}
			if (p2.size==0){
				p2.size++;
				p2.members[0]=p1.members[p1.size-1];
				p1.size--;
			}
			vnext=0;
		}

		(*t).label=-1;
		(*t).left=(dectree*)malloc(sizeof(dectree));
		(*t).left=generateTree(p1,g,vnext);
		(*t).right=(dectree*)malloc(sizeof(dectree));
		(*t).right=generateTree(p2,g,vnext);
	}
	return t;
}


pointset getCandidates (pointset left, pointset right, graph g){
	pointset cand;
	cand.size=0;
	cand.members=(int*)malloc(right.size*sizeof(int));
	pointset nleft;
	nleft.size=0;
	nleft.members=(int*)malloc((right.size+left.size)*sizeof(int));
	for (int j=0;j<right.size;j++){
		for (int i=0;i<left.size;i++){	
			if (g.matrix[left.members[i]*g.size+right.members[j]]==1){
				nleft.size++;
				nleft.members[nleft.size-1]=right.members[j];
				break;
			}
		}
	}

	for (int i=0;i<left.size;i++){
		nleft.size++;
		nleft.members[nleft.size-1]=left.members[i];
	}

	pointset nnleft;
	nnleft.size=0;
	nnleft.members=(int*)malloc((right.size+left.size)*sizeof(int));

	for (int i=0;i<right.size;i++){
		for (int j=0;j<nleft.size;j++){
			if (g.matrix[right.members[i]*g.size+nleft.members[j]]==1){
				nnleft.size++;
				nnleft.members[nnleft.size-1]=right.members[i];
				break;
			}
		}
	}

	return nnleft;
		
}


setwithinsets incrementun(graph g, pointset x, setwithinsets unx, int v){
	setwithinsets unv;
	unv.size=0;
	unv.set=(pointset*)malloc((g.size*g.size*g.size*g.size+g.size*g.size*g.size)*sizeof(pointset));

	for (int i=0; i<unx.size; i++){
		pointset s;
		s.size=0;
		s.members=(int*)malloc(g.size*sizeof(int));
	
		for (int j=0; j<unx.set[i].size; j++){
			if (unx.set[i].members[j]!=v){
				s.size++;
				s.members[s.size-1]=unx.set[i].members[j];
			}			
		}

		int alreadyin = 0;

		for (int j=0; j<unv.size; j++){
			if (s.size==unv.set[j].size){
				int common=0;
				for (int k=0;k<s.size;k++){
					for (int l=0;l<unv.set[j].size;l++){
						if (s.members[k]==unv.set[j].members[l]){
							common++;
							break;
						}
					}
				}
				if (common==s.size){
					alreadyin = 1;
					break;
				}
			}
		}
		if (alreadyin==0){
			unv.size++;
			unv.set[unv.size-1]=s;
		}

		pointset t;
		t.size=0;
		t.members=(int*)malloc(g.size*sizeof(int));

		for (int j=0; j<g.size; j++){
			if (j!=v){
				int inX=0;
				for (int k=0; k<x.size;k++){
					if(x.members[k]==j){
						inX=1;
						break;
					}
				}
				if (inX==0){
					if (g.matrix[j*g.size+v]==1){
						t.size++;
						t.members[t.size-1]=j;
					}
				}
			}
		}

		alreadyin = 0;

		for (int j=0; j<unv.size; j++){
			if (t.size==unv.set[j].size){
				int common=0;
				for (int k=0;k<t.size;k++){
					for (int l=0;l<unv.set[j].size;l++){
						if (t.members[k]==unv.set[j].members[l]){
							common++;
							break;
						}
					}
				}
				if (common==s.size){
					alreadyin = 1;
					break;
				}
			}
		}
		if (alreadyin==0){
			unv.size++;
			unv.set[unv.size-1]=t;
		}
		
	}

	return unv;
}


pointset incrementalUNheuristic (graph g, int init){

	pointset dec;
	dec.size=1;
	dec.members=(int*)malloc(g.size*sizeof(int));
	dec.members[0]=init;
	pointset left;
	left.size=1;
	left.members=(int*)malloc(g.size*sizeof(int));
	pointset right;
	right.size=0;
	right.members=(int*)malloc(g.size*sizeof(int));
	left.members[0]=init;

	for (int i=0;i<g.size;i++){
		if (i!=init){
			right.size++;
			right.members[right.size-1]=i;
		}
	}

	setwithinsets unleft;
	unleft.size=2;
	unleft.set=(pointset*)malloc((g.size*g.size*g.size*g.size+g.size*g.size*g.size)*sizeof(pointset));
	unleft.set[0].size=0;
	unleft.set[1].size=0;
	unleft.set[1].members=(int*)malloc(g.size*sizeof(int));
	
	for (int i=0; i<g.size;i++){
		if ((g.matrix[init*g.size+i]==1)||(i!=init)){
			unleft.set[1].size++;
			unleft.set[1].members[unleft.set[1].size-1]=i;
		}
	}

	while (right.size!=0){
		if (right.size==1){
			dec.size++;
			dec.members[dec.size-1]=right.members[0];
			right.size--;
		}
		else {
			pointset candidates = getCandidates (left, right, g);
			int chosen=-1;
			setwithinsets unchosen;
			unchosen.size=0;
			unchosen.set=(pointset*)malloc(200*(g.size*g.size*g.size*g.size+g.size*g.size*g.size)*sizeof(pointset));

			if (candidates.size==0){
				candidates=right;
			}
			for (int i=0; i<candidates.size; i++){
				setwithinsets unv;
				unv.size=0;
				unv.set=(pointset*)malloc((g.size*g.size*g.size*g.size+g.size*g.size*g.size)*sizeof(pointset));
				unv = incrementun(g, left, unleft, candidates.members[i]);
				if ((chosen==-1)||(unv.size<unchosen.size)){
					chosen=candidates.members[i];
					unchosen=unv;
				}
			}

			dec.size++;
			dec.members[dec.size-1]=chosen;
			left.size++;
			left.members[left.size-1]=chosen;
			pointset newright;
			newright.size=0;
			newright.members=(int*)malloc(g.size*sizeof(int));
			for (int i=0;i<right.size;i++){
				if (right.members[i]!=chosen){
					newright.size++;
					newright.members[newright.size-1]=right.members[i];
				}
			}
			right = newright;
			unleft=unchosen;
		}
	}

	return dec;
}


dectree *generateTreeBWstep (graph g, pointset dec, int i){
	dectree *t;
	t=(dectree*)malloc(sizeof(dectree));

	if (dec.size-i==0)
		return NULL;

	if (dec.size-1==i){
		(*t).label=dec.members[dec.size-i-1];
		(*t).right=NULL;
		(*t).left=NULL;
	}
	else {
		dectree *tleft;
		tleft=(dectree*)malloc(sizeof(dectree));
		dectree *tright;
		tright=(dectree*)malloc(sizeof(dectree));
		(*tright).label=dec.members[dec.size-i-1];
		(*tright).left=NULL;
		(*tright).right=NULL;
		tleft=generateTreeBWstep (g, dec, i+1);
		(*t).left=tleft;
		(*t).right=tright;
		(*t).label=-1; 
	}
	return t;
}


dectree *generateTreeBW (graph g){
	dectree **t = (dectree**)malloc(g.size*sizeof(dectree*));
	int *bw = (int*)malloc(g.size*sizeof(int));
	for (int i=0; i<g.size; i++){
		pointset dec=incrementalUNheuristic (g, i);
		t[i]=(dectree*)malloc(sizeof(dectree));
		dectree *tleft;
		tleft=(dectree*)malloc(sizeof(dectree));
		dectree *tright;
		tright=(dectree*)malloc(sizeof(dectree));
		(*tright).label=dec.members[dec.size-1];
		(*tright).left=NULL;
		(*tright).right=NULL;
		tleft=generateTreeBWstep (g, dec, 1);
		t[i]->left=tleft;
		t[i]->right=tright;
		t[i]->label=-1;
		bw[i]=getBW(t[i]->left, &g);
		bw[i]+=getBW(t[i]->right, &g);
	} 

	int size=bw[0];
	int min=0;

	for (int i=1; i<g.size; i++){
		if (bw[i]<size){
			size=bw[i];
			min=i;
		}
	}
	return t[min];
}


int printTree (dectree t){
	printf("Un arbre");
	if (t.label!=-1)
		printf(" de label %d.", t.label);
	else {
		if (t.left!=NULL){
			printf(" dont le fils gauche est {");
			printTree(*(t.left));
			printf("}");
		}
		if (t.right != NULL){
			printf(" et dont le fils droit est {");
			printTree(*(t.right));
			printf("}");
		}
	}
	printf("\n");
	return EXIT_SUCCESS;
}
