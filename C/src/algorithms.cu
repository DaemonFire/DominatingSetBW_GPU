#include "hip/hip_runtime.h"
#include "../include/algorithms.h"
#include "../include/treeprimitives.h"


#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <errno.h>
#include <sys/stat.h>
#include <pthread.h>

int* nnodes;
dectree*** nodestocompute;
int* tocompute;
pthread_mutex_t* mut;
graph** gwork;

int inital (int ncomp){
	nnodes = (int*)malloc(ncomp*sizeof(int));
	nodestocompute = (dectree***)malloc(ncomp*sizeof(dectree**));
	tocompute = (int*)malloc(ncomp*sizeof(int));
	mut = (pthread_mutex_t*)malloc(ncomp*sizeof(pthread_mutex_t));
	gwork = (graph**)malloc(ncomp*sizeof(graph*));
	for (int i=0; i<ncomp; i++)
		pthread_mutex_init(&mut[i], NULL);
	return EXIT_SUCCESS;
}

int fillThevoid (dectree* t, graph* g, int i){
	gwork[i]=g;
	t->computed=0;
	nodestocompute[i][tocompute[i]]=t;
	tocompute[i]++;
	if (t->left!=NULL)
		fillThevoid(t->left, g, i);
	if (t->right!=NULL)
		fillThevoid(t->right, g, i);
	return EXIT_SUCCESS;
}

int getnumberofnodes(dectree* t){
	int n=1;
	if (t->left!=NULL)
		n+=getnumberofnodes(t->left);
	if (t->right!=NULL)
		n+=getnumberofnodes(t->right);
	return n;
}

__global__
void computeTwins (int* mat, int* width, int* result){
	int twin=1;
	for (int i=0; i<*width; i++){
		if (mat[blockIdx.x*(*width) + i]!=mat[threadIdx.x*(*width)+i])
			twin = 0;
	}
	if (twin==1)
		result[threadIdx.x*blockDim.x+blockIdx.x]=1;
}

__global__
void computeRepresentatives(int* result, int* ptr){
	for (int i=0; i<blockDim.x; i++){
		if (result[threadIdx.x*blockDim.x+i]==1){
			ptr[threadIdx.x*2]=threadIdx.x;
			if (i<threadIdx.x)
				ptr[threadIdx.x*2+1]=i;
			else
				ptr[threadIdx.x*2+1]=threadIdx.x;
			break;
		}
	}
}
	
__global__
void computeNeighboorhoods(int* lc, int* rc, int* lrc, int* lrcard, int* mat, int* width, int* na, int* nacomp, int*a, int* acomp, int* rep, int* repc){
	for (int i = 0; i<*width; i++){
		int indexi=-1;
		for (int k=0; k<(*nacomp); k++){
			if (acomp[k]==repc[i]){
				indexi=k;
				break;	
			}
		}
		int isin = 0;
		for (int j = 0 ; j<blockDim.x; j++){
			int indexj=-1;
			for (int k=0; k<(*na); k++){
				if (a[k]==rep[j]){
					indexj=k;
					break;
				}
			}
			if ((lc[(blockIdx.x*blockDim.x+threadIdx.x)*blockDim.x+j]==1)&&(mat[indexj*(*nacomp)+indexi]==1)){
				isin=1;
				break;
			}
		}
		if (isin==1)
			rc[(blockIdx.x*blockDim.x+threadIdx.x)*(*width)+i]=1;
		else
			rc[(blockIdx.x*blockDim.x+threadIdx.x)*(*width)+i]=0;
	}
	
}

__global__
void computeAlgorithm (int* tabg, int* lra, int* lrb, int* lrwc, int* lracard, int* lrbcard, int* lrwcard, int* lnracard, int* lnrbcard, int* lnrwcard, int* mw, int* macomp, int* mbcomp, int* nrepac, int* nrepbc, int* nrepwc, int* nrepa, int* nrepb, int* nrepw, int* repacomp, int* repbcomp, int* repwcomp, int* repa, int* repb, int* repw, int* taba, int* tabb, int* ptrac, int* ptrbc, int* ptrw, int* nacomp, int* nbcomp, int* nw){
	int indexa = threadIdx.x/(*lrbcard);
	int indexb = threadIdx.x%(*lrbcard);
	int indexbc = 0;
	int indexac = 0;
	int indexw = 0;

	for (int i = 0; i<(*nrepwc); i++){
		if (lrwc[blockIdx.x*(*nrepwc)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nbcomp); j++){
				if (ptrbc[2*j]==repwcomp[i]){
					rep=ptrbc[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepbc); j++){
				if (repbcomp[j]==rep){
					rep=j;
					break;
				}
			}
			indexbc = mbcomp[indexbc*(*nrepbc)+rep];
		}
	}
	for (int i = 0; i<(*nrepa); i++){
		if (lra[indexa*(*nrepa)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nbcomp); j++){
				if (ptrbc[2*j]==repa[i]){
					rep=ptrbc[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepbc); j++){
				if (repbcomp[j]==rep){
					rep=j;
					break;
				}
			}
			indexbc = mbcomp[indexbc*(*nrepbc)+rep];
		}
	}
	for (int i = 0; i<(*nrepwc); i++){
		if (lrwc[blockIdx.x*(*nrepwc)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nacomp); j++){
				if (ptrac[2*j]==repwcomp[i]){
					rep=ptrac[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepac); j++){
				if (repacomp[j]==rep){
					rep=j;
					break;
				}
			}
			indexac = macomp[indexac*(*nrepac)+rep];
		}
	}
	for (int i = 0; i<(*nrepb); i++){
		if (lrb[indexb*(*nrepb)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nacomp); j++){
				if (ptrac[2*j]==repb[i]){
					rep=ptrac[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepac); j++){
				if (repacomp[j]==rep){
					rep=j;
					break;
				}
			}
			indexac = macomp[indexac*(*nrepac)+rep];
		}
	}
	for (int i = 0; i<(*nrepa); i++){
		if (lra[indexa*(*nrepa)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nw); j++){
				if (ptrw[2*j]==repa[i]){
					rep=ptrw[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepw); j++){
				if (repw[j]==rep){
					rep=j;
					break;
				}
			}
			indexw = mw[indexw*(*nrepw)+rep];
		}
	}
	for (int i = 0; i<(*nrepb); i++){
		if (lrb[indexb*(*nrepb)+i]==1){
			int rep = -1;
			for (int j=0; j< (*nw); j++){
				if (ptrw[2*j]==repb[i]){
					rep=ptrw[2*j+1];
					break;
				}
			}
			for (int j = 0; j< (*nrepw); j++){
				if (repw[j]==rep){
					rep=j;
					break;
				}
			}
			indexw = mw[indexw*(*nrepw)+rep];
		}
	}

	tabg[5*blockDim.x*blockIdx.x+5*threadIdx.x]=indexw;
	tabg[5*blockDim.x*blockIdx.x+5*threadIdx.x+1]=indexac;
	tabg[5*blockDim.x*blockIdx.x+5*threadIdx.x+2]=indexbc;
	tabg[5*blockDim.x*blockIdx.x+5*threadIdx.x+3]=taba[indexa*(*lnracard)+indexac];
	tabg[5*blockDim.x*blockIdx.x+5*threadIdx.x+4]=tabb[indexb*(*lnrbcard)+indexbc];
}

cutdata cutThatTree (graph* g, dectree* t){
	cutdata c;
	c.na=0;
	c.nacomp=0;
	c.a=NULL;
	c.acomp=NULL;

	c.na=getnumberofleaves (*(t));
	c.a=(int*)malloc(c.na*sizeof(int));
	getallleaves(*(t), c.a);
	
	c.nacomp=g->size-c.na;

	c.acomp=(int*)malloc(c.nacomp*sizeof(int));

	int i=0;
	int j=0;
	int k=0;
	for (i=0;i<g->size;i++){
		int ina=0;
		for (j=0;j<c.na;j++){
			if (c.a[j]==i){
				ina=1;
				break;
			}
		}
		if (ina==0){
			c.acomp[k]=i;
			k++;
			if (k==c.nacomp)
				break;
		}
	}

	c.matrixrevisited = (int*)malloc(c.na*c.nacomp*sizeof(int));


	for (int i=0;i<c.na;i++){
		for (int j=0;j<c.nacomp;j++)
			c.matrixrevisited[i*c.nacomp+j]=g->matrix[c.a[i]*g->size+c.acomp[j]];
	}
	return c;
}


__global__ 	
void computeMatrix(int* lr, int* ln, int* lrcard, int* tc, int* mg, int* mat, int* na, int* nacomp, int* a, int* acomp, int* rep, int* repc, int* width, int* r){
	int index= -1;
	for (int i=0; i<(*na);i++){
		if (a[i]==rep[threadIdx.x]){
			index=i;
			break;
		}
	}
	for (int i = 0; i <*width; i++){
		int indexi=-1;
		for (int j=0; j<(*nacomp);j++){
			if (acomp[j]==repc[i]){
				indexi=j;
				break;
			}
		}
		int isin=0;
		if (mat[index*(*nacomp)+indexi]==1)
			isin = 1;
		if (isin==0){
			for (int j = 0; j<blockDim.x; j++){
				int indexj=-1;
				for (int k=0; k<(*na);k++){
					if (a[k]==rep[j]){
						indexj=k;
						break;
					}
				}
				if ((lr[blockIdx.x*blockDim.x+j]==1)&&(mat[indexj*(*nacomp)+indexi]==1)){
					isin = 1;
					break;
				}
			}
		}		
		if (isin==1)
			r[(blockIdx.x*blockDim.x+threadIdx.x)*(*width)+i]=1;
		else
			r[(blockIdx.x*blockDim.x+threadIdx.x)*(*width)+i]=0;
	}
	int answer=-1;
	for (int i = 0; i <(*lrcard); i++){
		int id = 1;
		for (int j = 0; j<(*width); j++){
			if (r[(blockIdx.x*blockDim.x+threadIdx.x)*(*width)+j]!=ln[i*(*width)+j]){
				id = 0;
				break;
			}
		}
		if (id==1){
			answer=i;
			break;
		}
	}
	mg[blockIdx.x*blockDim.x+threadIdx.x]=answer;
}

int firstpreprocess(graph* g,  cutdata* c){

	int* res;

	res= (int*) malloc(c->na*c->na*sizeof(int));

	for (int i=0; i<c->na*c->na; i++)
		res[i]=0;

	int* mat;
	int* width;
	int * result;

	hipMalloc((void**)&mat, c->na*c->nacomp*sizeof(int));
	hipMalloc((void**)&width, sizeof(int));
	hipMalloc((void**)&result, c->na*c->na*sizeof(int));
	
	hipMemcpy(result, res, c->na*c->na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mat, c->matrixrevisited, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &(c->nacomp), sizeof(int), hipMemcpyHostToDevice);

	computeTwins<<<c->na,c->na>>>(mat, width, result);

	hipMemcpy(res, result, c->na*c->na*sizeof(int), hipMemcpyDeviceToHost);

	c->pointtorep=(int*)malloc(2*c->na*sizeof(int));
	c->pointtorepincomp=(int*)malloc(2*c->nacomp*sizeof(int));
	
	for (int i=0; i<2*c->na;i++){
		c->pointtorep[i]=-1;
	}

	for (int i=0;i<2*c->nacomp;i++){
		c->pointtorepincomp[i]=-1;
	}

	int* ptr;
	
	hipMalloc((void**)&ptr, 2*c->na*sizeof(int));

	hipMemcpy(ptr, c->pointtorep, 2*c->na*sizeof(int), hipMemcpyHostToDevice);

	computeRepresentatives<<<1,c->na>>>(result, ptr);	
	
	hipMemcpy(c->pointtorep, ptr, 2*c->na*sizeof(int), hipMemcpyDeviceToHost);


	int* res2= (int*) malloc(c->nacomp*c->nacomp*sizeof(int));
	int* result2;

	int *reversedMatrix = (int*)malloc(c->na*c->nacomp*sizeof(int));
	for (int i =0; i<c->na; i++){
		for (int j=0; j<c->nacomp; j++){
			reversedMatrix[j*c->na+i]=c->matrixrevisited[i*c->nacomp+j];
		}
	}

	for (int i = 0; i< c-> nacomp*c->nacomp;i++)
		res2[i]=0;

	hipMalloc((void**)&result2, c->nacomp*c->nacomp*sizeof(int));

	hipMemcpy(result2, res2, c->nacomp*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mat, reversedMatrix, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &(c->na), sizeof(int), hipMemcpyHostToDevice);

	computeTwins<<<c->nacomp,c->nacomp>>>(mat, width, result2);

	int* ptr2;

	hipMalloc((void**)&ptr2, 2*c->nacomp*sizeof(int));

	hipMemcpy(ptr2, c->pointtorepincomp, 2*c->nacomp*sizeof(int), hipMemcpyHostToDevice);

	computeRepresentatives<<<1,c->nacomp>>>(result2, ptr2);

	hipMemcpy(c->pointtorepincomp, ptr2, 2*c->nacomp*sizeof(int), hipMemcpyDeviceToHost);
		
	for (int i = 0; i< 2*c->na; i++)
		c->pointtorep[i]=c->a[c->pointtorep[i]];
	for (int i = 0; i< 2*c->nacomp; i++)
		c->pointtorepincomp[i]=c->acomp[c->pointtorepincomp[i]];



	c->tc=(int*)malloc(c->na*sizeof(int));
	for (int i=0;i<c->na;i++)
		c->tc[i]=-1;
	c->complementtc=(int*)malloc(c->nacomp*sizeof(int));
	int cursor = 0;
	for (int i=0;i<c->na;i++){
		int here=0;
		for (int j=0;j<c->na;j++){
			if (c->pointtorep[2*i+1]==c->tc[j]){
				here=1;
				break;
			}
		}
		if (here==0){
			c->tc[cursor]=c->pointtorep[2*i+1];			
			cursor++;
		}
	}

	c->nrep=cursor;

	for (int i=0;i<c->nacomp;i++)
		c->complementtc[i]=-1;

	cursor=0;
	for (int i=0;i<c->nacomp;i++){
		int here=0;
		for (int j=0;j<c->nacomp;j++){
			if (c->pointtorepincomp[2*i+1]==c->complementtc[j]){
				here=1;
				break;
			}
		}
		if (here==0){
			c->complementtc[cursor]=c->pointtorepincomp[2*i+1];
			cursor++;
		}
	}
	c->nrepincomp=cursor;

	return EXIT_SUCCESS;

}



int secondpreprocess (cutdata* c, graph* g){


	c->lra = (int*) malloc (c->nrep*sizeof(int));
	c->lnra = (int*) malloc (c->nrepincomp*sizeof(int));
	c->lracard=1;
	c->lnracard=1;

	for (int i = 0; i<c->nrep; i++)
	c->lra[i]=0;
	for (int i = 0; i< c->nrepincomp; i++)
	c->lnra[i]=0;

	int *nextLevel;
	int *lastLevel=(int*)malloc(c->nrep*sizeof(int));
	for (int i=0; i<c->nrep; i++)
		lastLevel[i]=0;

	int sizeoflast=1;
	int sizeofnext=0;

	while (sizeoflast!=0){
		int* l = (int*)malloc(sizeoflast*c->nrep*c->nrep*sizeof(int));
		int* r = (int*)malloc(sizeoflast*c->nrep*c->nrepincomp*sizeof(int));

		for (int i = 0; i< sizeoflast; i ++){
			for (int j = 0; j<c->nrep; j++) {
				for (int k=0; k<c->nrep; k++){
					if (k==j)
						l[(i*c->nrep+j)*c->nrep+k]=1;
					else
						l[(i*c->nrep+j)*c->nrep+k]=lastLevel[i*c->nrep+k];
				}
			}
		}
		int* lc;
		int* rc;
		int* lrc;
		int* lrcard;
		int* mat;
		int* width;
		int* na;
		int* nacomp;
		int* a;
		int* acomp;
		int* rep;
		int* repc;


		hipMalloc((void**)&lc, sizeoflast*c->nrep*c->nrep*sizeof(int));
		hipMalloc((void**)&rc, sizeoflast*c->nrep*c->nrepincomp*sizeof(int));
		hipMalloc((void**)&lrc, c->lnracard*c->nrepincomp*sizeof(int));
		hipMalloc((void**)&mat, c->na * c-> nacomp * sizeof(int));
		hipMalloc((void**)&lrcard, sizeof(int));
		hipMalloc((void**)&width, sizeof(int));
		hipMalloc((void**)&na, sizeof(int));
		hipMalloc((void**)&nacomp, sizeof(int));
		hipMalloc((void**)&a, c->na*sizeof(int));
		hipMalloc((void**)&acomp, c->nacomp*sizeof(int));
		hipMalloc((void**)&rep, c->nrep*sizeof(int));
		hipMalloc((void**)&repc, c->nrepincomp*sizeof(int));

		hipMemcpy(lc, l, sizeoflast*c->nrep*c->nrep*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(rc, r, sizeoflast*c->nrep*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(lrc, c->lnra, c->lnracard*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(lrcard, &(c->lnracard), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(mat, c->matrixrevisited, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(width, &(c->nrepincomp), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(na, &(c->na), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(nacomp, &(c->nacomp), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(a, c->a, c->na*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(acomp, c->acomp, c->nacomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(rep, c->tc, c->nrep*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(repc, c->complementtc, c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);

		computeNeighboorhoods<<<sizeoflast,c->nrep>>>(lc, rc, lrc, lrcard, mat, width, na, nacomp, a, acomp, rep, repc);

		hipMemcpy(l, lc, sizeoflast*c->nrep*c->nrep*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(r, rc, sizeoflast*c->nrep*c->nrepincomp*sizeof(int), hipMemcpyDeviceToHost);
		int *ltemp = (int*)malloc((c->lracard+sizeoflast*c->nrep)*c->nrep*sizeof(int));
		int *lrtemp = (int*)malloc((c->lnracard+sizeoflast*c->nrep)*c->nrepincomp*sizeof(int));
		nextLevel = (int*)malloc(sizeoflast*c->nrep*c->nrep*sizeof(int));
		


		for (int i= 0; i< c->lracard*c->nrep;i++)
			ltemp[i]=c->lra[i];
		for (int i = 0; i<c->lnracard*c->nrepincomp;i++)
			lrtemp[i]=c->lnra[i];
		sizeofnext = 0;
		for (int i = 0; i < sizeoflast*c->nrep; i++){
			if (r[i*c->nrepincomp]!=-1){
				int alreadyin = 0;
				for (int j=0; j<c->lnracard+sizeofnext; j++){
					int id=1;
					for (int k= 0; k<c->nrepincomp; k++){
						if (r[i*c->nrepincomp+k]!=lrtemp[j*c->nrepincomp+k]){
							id=0;
							break;
						}
					}
					if (id==1){
						alreadyin=1;
						break;
					}
				}
				if (alreadyin ==0){
					for (int j =0; j<c->nrep; j++){
						nextLevel[sizeofnext*c->nrep+j]=l[i*c->nrep+j];
						ltemp[(c->lracard+sizeofnext)*c->nrep+j]=l[i*c->nrep+j];
					}
					for (int j =0; j<c->nrepincomp; j++){
						lrtemp[(c->lnracard+sizeofnext)*c->nrepincomp+j]=r[i*c->nrepincomp+j];
					}
					sizeofnext++;
				}
			}
		}
		c->lracard=c->lracard+sizeofnext;
		c->lnracard=c->lnracard+sizeofnext;
		c->lra=ltemp;
		c->lnra=lrtemp;



		lastLevel=nextLevel;
		sizeoflast=sizeofnext;
	}

	c->lracomp = (int*) malloc (c->nrepincomp*sizeof(int));
	c->lnracomp = (int*) malloc (c->nrep*sizeof(int));
	c->lracompcard=1;
	c->lnracompcard=1;

	for (int i = 0; i<c->nrepincomp; i++)
	c->lracomp[i]=0;
	for (int i = 0; i< c->nrep; i++)
	c->lnracomp[i]=0;

	nextLevel=NULL;
	lastLevel=NULL;
	lastLevel=(int*)malloc(c->nrepincomp*sizeof(int));
	for (int i=0; i<c->nrepincomp; i++)
		lastLevel[i]=0;

	sizeoflast=1;
	sizeofnext=0;

	while (sizeoflast!=0){
		int* l = (int*)malloc(sizeoflast*c->nrepincomp*c->nrepincomp*sizeof(int));
		int *r = (int*)malloc(sizeoflast*c->nrep*c->nrepincomp*sizeof(int));

		for (int i = 0; i< sizeoflast; i ++){
			for (int j = 0; j<c->nrepincomp; j++) {
				for (int k=0; k<c->nrepincomp; k++){
					if (k==j)
						l[(i*c->nrepincomp+j)*c->nrepincomp+k]=1;
					else
						l[(i*c->nrepincomp+j)*c->nrepincomp+k]=lastLevel[i*c->nrepincomp+k];
				}
			}
		}
		int* lc;
		int* rc;
		int* lrc;
		int* lrcard;
		int* mat;
		int* width;
		int* na;
		int* nacomp;
		int* a;
		int* acomp;
		int* rep;
		int* repc;
		int* revMatrix = (int*)malloc(c->na*c->nacomp*sizeof(int));

		for (int i = 0; i<c->na; i++){
			for (int j =0 ;j <c->nacomp; j++){
				revMatrix[j*c->na+i]=c->matrixrevisited[i*c->nacomp+j];
			}
		}


		hipMalloc((void**)&lc, sizeoflast*c->nrepincomp*c->nrepincomp*sizeof(int));
		hipMalloc((void**)&rc, sizeoflast*c->nrepincomp*c->nrep*sizeof(int));
		hipMalloc((void**)&lrc, c->lnracompcard*c->nrep*sizeof(int));
		hipMalloc((void**)&mat, c->nacomp * c-> na * sizeof(int));
		hipMalloc((void**)&lrcard, sizeof(int));
		hipMalloc((void**)&width, sizeof(int));
		hipMalloc((void**)&na, sizeof(int));
		hipMalloc((void**)&nacomp, sizeof(int));
		hipMalloc((void**)&a, c->nacomp*sizeof(int));
		hipMalloc((void**)&acomp, c->na*sizeof(int));
		hipMalloc((void**)&rep, c->nrepincomp*sizeof(int));
		hipMalloc((void**)&repc, c->nrep*sizeof(int));

		hipMemcpy(lc, l, sizeoflast*c->nrepincomp*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(rc, r, sizeoflast*c->nrep*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(lrc, c->lnracomp, c->lnracompcard*c->nrep*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(lrcard, &(c->lnracompcard), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(mat, revMatrix, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(width, &(c->nrep), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(na, &(c->nacomp), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(nacomp, &(c->na), sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(a, c->acomp, c->nacomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(acomp, c->a, c->na*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(rep, c->complementtc, c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(repc, c->tc, c->nrep*sizeof(int), hipMemcpyHostToDevice);

		computeNeighboorhoods<<<sizeoflast,c->nrepincomp>>>(lc, rc, lrc, lrcard, mat, width, na, nacomp, a, acomp, rep, repc);

		hipMemcpy(l, lc, sizeoflast*c->nrepincomp*c->nrepincomp*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(r, rc, sizeoflast*c->nrep*c->nrepincomp*sizeof(int), hipMemcpyDeviceToHost);
		int *ltemp = (int*)malloc((c->lracompcard+sizeoflast*c->nrepincomp)*c->nrepincomp*sizeof(int));
		int *lrtemp = (int*)malloc((c->lnracompcard+sizeoflast*c->nrepincomp)*c->nrep*sizeof(int));
		nextLevel = (int*)malloc(sizeoflast*c->nrepincomp*c->nrepincomp*sizeof(int));

		for (int i= 0; i< c->lracompcard*c->nrepincomp;i++)
			ltemp[i]=c->lracomp[i];
		for (int i = 0; i<c->lnracompcard*c->nrep;i++)
			lrtemp[i]=c->lnracomp[i];
		
		sizeofnext = 0;
		for (int i = 0; i < sizeoflast*c->nrepincomp; i++){
			if (r[i*c->nrep]!=-1){
				int alreadyin = 0;
				for (int j=0; j<c->lnracompcard+sizeofnext; j++){
					int id=1;
					for (int k= 0; k<c->nrep; k++){
						if (r[i*c->nrep+k]!=lrtemp[j*c->nrep+k]){
							id=0;
							break;
						}
					}
					if (id==1){
						alreadyin=1;
						break;
					}
				}
				if (alreadyin ==0){
					for (int j =0; j<c->nrepincomp; j++){
						nextLevel[sizeofnext*c->nrepincomp+j]=l[i*c->nrepincomp+j];
						ltemp[(c->lracompcard+sizeofnext)*c->nrepincomp+j]=l[i*c->nrepincomp+j];
					}
					for (int j =0; j<c->nrep; j++){
						lrtemp[(c->lnracompcard+sizeofnext)*c->nrep+j]=r[i*c->nrep+j];
					}
					sizeofnext++;
				}
			}
		}
		c->lracompcard=c->lracompcard+sizeofnext;
		c->lnracompcard=c->lnracompcard+sizeofnext;
		c->lracomp=ltemp;
		c->lnracomp=lrtemp;
		lastLevel=nextLevel;
		sizeoflast=sizeofnext;
		
	}



	return EXIT_SUCCESS;
}


int thirdpreprocess (cutdata* c, graph* g){

	c->m=(int*)malloc(c->lracard*c->nrep*sizeof(int));
	for (int i=0; i<c->lracard*c->nrep; i++)
		c->m[i]=-1;
	int* lr;
	int* ln;
	int* lrcard;
	int* tc;
	int* mg;
	int* mat;
	int* na;
	int* nacomp;
	int* a;
	int* acomp;
	int* rep;
	int* repc;
	int* width;
	int* r;
	

	hipMalloc((void**)&lr, c->lracard*c->nrep*sizeof(int));
	hipMalloc((void**)&ln, c->lnracard*c->nrepincomp*sizeof(int));
	hipMalloc((void**)&lrcard, sizeof(int));
	hipMalloc((void**)&tc, c->nrep*sizeof(int));
	hipMalloc((void**)&mg, c->lracard*c->nrep*sizeof(int));
	hipMalloc((void**)&mat, c->na*c->nacomp*sizeof(int));
	hipMalloc((void**)&na, sizeof(int));
	hipMalloc((void**)&nacomp, sizeof(int));
	hipMalloc((void**)&a, c->na*sizeof(int));
	hipMalloc((void**)&acomp, c->nacomp*sizeof(int));
	hipMalloc((void**)&rep, c->nrep*sizeof(int));
	hipMalloc((void**)&repc, c->nrepincomp*sizeof(int));
	hipMalloc((void**)&width, sizeof(int));	
	hipMalloc((void**)&r, c->lracard*c->nrep*c->nrepincomp*sizeof(int));

	hipMemcpy(lr, c->lra, c->lracard*c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ln, c->lnra, c->lnracard*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(lrcard, &(c->lnracard), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(tc, c->tc, c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mg, c->m, c->lracard*c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mat, c->matrixrevisited, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(na, &(c->na), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nacomp, &(c->nacomp), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a, c->a, c->na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(acomp, c->acomp, c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rep, c->tc, c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(repc, c->complementtc, c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &(c->nrepincomp), sizeof(int), hipMemcpyHostToDevice);
	
	computeMatrix<<<c->lracard, c->nrep>>>(lr, ln, lrcard, tc, mg, mat, na, nacomp, a, acomp, rep, repc, width, r);

	hipMemcpy(c->m, mg, c->lracard*c->nrep*sizeof(int), hipMemcpyDeviceToHost);

	c->mcomp=(int*)malloc(c->lracompcard*c->nrepincomp*sizeof(int));
	for (int i=0; i<c->lracompcard*c->nrepincomp; i++)
		c->mcomp[i]=-1;

	hipMalloc((void**)&lr, c->lracompcard*c->nrepincomp*sizeof(int));
	hipMalloc((void**)&ln, c->lnracompcard*c->nrep*sizeof(int));
	hipMalloc((void**)&lrcard, sizeof(int));
	hipMalloc((void**)&tc, c->nrepincomp*sizeof(int));
	hipMalloc((void**)&mg, c->lracompcard*c->nrepincomp*sizeof(int));
	hipMalloc((void**)&mat, c->nacomp*c->na*sizeof(int));
	hipMalloc((void**)&na, sizeof(int));
	hipMalloc((void**)&nacomp, sizeof(int));
	hipMalloc((void**)&a, c->nacomp*sizeof(int));
	hipMalloc((void**)&acomp, c->na*sizeof(int));
	hipMalloc((void**)&rep, c->nrepincomp*sizeof(int));
	hipMalloc((void**)&repc, c->nrep*sizeof(int));
	hipMalloc((void**)&width, sizeof(int));
	hipMalloc((void**)&r, c->lracompcard*c->nrepincomp*c->nrep*sizeof(int));

	int* reversedMatrix = (int*) malloc(c->na*c->nacomp*sizeof(int));
	for (int i = 0; i<c->na; i++){
		for (int j=0; j<c->nacomp; j++){
			reversedMatrix[j*c->na+i]=c->matrixrevisited[i*c->nacomp+j];
		}
	}
	

	hipMemcpy(lr, c->lracomp, c->lracompcard*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ln, c->lnracomp, c->lnracompcard*c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(lrcard, &(c->lnracompcard), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(tc, c->complementtc, c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mg, c->mcomp, c->lracompcard*c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mat, reversedMatrix, c->na*c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(na, &(c->nacomp), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nacomp, &(c->na), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a, c->acomp, c->nacomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(acomp, c->a, c->na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(rep, c->complementtc, c->nrepincomp*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(repc, c->tc, c->nrep*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &(c->nrep), sizeof(int), hipMemcpyHostToDevice);

	computeMatrix<<<c->lracompcard, c->nrepincomp>>>(lr, ln, lrcard, tc, mg, mat, na,nacomp, a, acomp, rep, repc, width, r);

	hipMemcpy(c->mcomp, mg, c->lracompcard*c->nrepincomp*sizeof(int), hipMemcpyDeviceToHost);

	return EXIT_SUCCESS;
}

void *threadAlgorithm ( void *arg){
	int j = *((int*)arg);
	int i=0;
	int treating=1;
	while (treating>0){
		pthread_mutex_lock(&mut[j]);
		tocompute[j]=nnodes[j];
		for (int k=0 ; k<nnodes[j]; k++){
			if (nodestocompute[j][k]->computed==1)
				tocompute[j]--;
		}
		if (tocompute[j]<=0){
			pthread_mutex_unlock(&mut[j]);
			treating=0;
		}
		else {
			if (nodestocompute[j][i]->computed==0){
			//	printf("Thread %d computing %d\n", j, i);
				nodestocompute[j][i]->computed=2;
			/*	printf("State of the art with to compute at %d\n", tocompute);
				for (int k=0; k<nnodes; k++)
					printf("node %d is computed=%d\n", k, nodestocompute[k]->computed);*/
				pthread_mutex_unlock(&mut[j]);
				stepalgorithm(nodestocompute[j][i],gwork[j]);
			}
			else{
			/*	printf("State of the art with to compute at %d\n", tocompute);
				for (int k=0; k<nnodes; k++)
					printf("node %d is computed=%d\n", k, nodestocompute[k]->computed);*/
				pthread_mutex_unlock(&mut[j]);
			}
			i=(i+1)%nnodes[j];
			
		}

	}
	printf("Thread closing\n");
	pthread_exit(NULL);
}

int toplevelalgorithm (dectree* t, graph* g, int n, int* set, int z){

	if ((t->right==NULL)||(t->left==NULL)){
		return 0;
	}

	nnodes[z]=getnumberofnodes(t)-1;
	tocompute[z]=0;
	nodestocompute[z]=(dectree**)malloc(nnodes[z]*sizeof(dectree*));
	if (t->right!=NULL)
		fillThevoid(t->right,g, z);
	else
		return toplevelalgorithm(t->left, g, n, set, z);

	if (t->left!=NULL)
		fillThevoid(t->left, g, z);
	
	pthread_t* threads = (pthread_t*)malloc(n*sizeof(pthread_t));

	for (int i=0; i<n; i++){
		if (pthread_create(&threads[i], NULL, threadAlgorithm, &z)){
			perror("pthread_create");
			return 0;
		}
	}
	printf("Initiating threads\n");

	int finished=0;
	while (finished==0){
		pthread_mutex_lock(&mut[z]);
		tocompute[z] = nnodes[z];
		for (int i=0; i<nnodes[z]; i++){
			if (nodestocompute[z][i]->computed==1)
				tocompute[z]--;
		}	
	//	printf("%d / %d\n\n\n", tocompute, nnodes);
		if (tocompute[z]<=0)
			finished=1;
		pthread_mutex_unlock(&mut[z]);
	}
	printf("-----------------------------------------------Preparing to close\n");
	for (int i=0; i<n; i++)
		pthread_join(threads[i],NULL);
	//sleep(4);
	printf("--------------------------------------------------------closing\n");
/*
	for (int i =0; i<nnodes[z]; i++){
		printf("Data of node %d-----------------------\n",i);
		printf("a = ");
		for (int j=0; j<nodestocompute[z][i]->c.na; j++)
			printf("%d, ",nodestocompute[z][i]->c.a[j]);
		printf("\n");
		printf("acomp = ");
		for (int j=0; j<nodestocompute[z][i]->c.nacomp; j++)
			printf("%d, ", nodestocompute[z][i]->c.acomp[j]);
		printf("\n");
		printf("tc = ");
		for (int j=0; j<nodestocompute[z][i]->c.nrep; j++)
			printf("%d, ", nodestocompute[z][i]->c.tc[j]);
		printf("\n");
		printf("complementtc = ");
		for (int j=0; j<nodestocompute[z][i]->c.nrepincomp; j++)
			printf("%d, ", nodestocompute[z][i]->c.complementtc[j]);
		printf("\n");
		printf("pointtorep = \n");
		for (int j=0; j<nodestocompute[z][i]->c.na; j++)
			printf("%d->%d\n", nodestocompute[z][i]->c.pointtorep[2*j], nodestocompute[z][i]->c.pointtorep[2*j+1]);
		printf("pointtorepincomp = \n");
		for (int j=0; j<nodestocompute[z][i]->c.nacomp; j++)
			printf("%d->%d\n", nodestocompute[z][i]->c.pointtorepincomp[2*j], nodestocompute[z][i]->c.pointtorepincomp[2*j+1]);
		printf("matrixrevisited = \n");
		for (int j=0; j<nodestocompute[z][i]->c.na; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nacomp; k++)
				printf("%d ", nodestocompute[z][i]->c.matrixrevisited[j*nodestocompute[z][i]->c.nacomp+k]);
			printf("\n");
		}
		printf("lra = \n");
		for (int j=0; j<nodestocompute[z][i]->c.lracard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrep; k++)
				printf("%d, ", nodestocompute[z][i]->c.lra[j*nodestocompute[z][i]->c.nrep+k]);
			printf("\n");
		}
		printf("lnra = \n");
		for (int j=0; j<nodestocompute[z][i]->c.lnracard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrepincomp; k++)
				printf("%d, ", nodestocompute[z][i]->c.lnra[j*nodestocompute[z][i]->c.nrepincomp+k]);
			printf("\n");
		}
		printf("lracomp = \n");
		for (int j=0; j<nodestocompute[z][i]->c.lracompcard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrepincomp;k++)
				printf("%d, " , nodestocompute[z][i]->c.lracomp[j*nodestocompute[z][i]->c.nrepincomp+k]);
			printf("\n");
		}
		printf("lnracomp = \n");
		for (int j=0; j<nodestocompute[z][i]->c.lnracompcard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrep; k++)
				printf("%d, ", nodestocompute[z][i]->c.lnracomp[j*nodestocompute[z][i]->c.nrep+k]);
			printf("\n");
		}
		printf("m =\n");
		for (int j=0; j<nodestocompute[z][i]->c.lracard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrep; k++)
				printf("%d ", nodestocompute[z][i]->c.m[j*nodestocompute[z][i]->c.nrep+k]);
			printf("\n");
		}
		printf("mcomp =\n");
		for (int j=0; j<nodestocompute[z][i]->c.lracompcard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.nrepincomp; k++)
				printf("%d ", nodestocompute[z][i]->c.mcomp[j*nodestocompute[z][i]->c.nrepincomp+k]);
			printf("\n");
		}
		printf("tab =\n");
		for (int j=0; j<nodestocompute[z][i]->c.lracard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.lracompcard; k++){
				printf("%d ",nodestocompute[z][i]->c.tab[j*nodestocompute[z][i]->c.lracompcard+k]);
			}
			printf("\n");
		}

		printf("box =\n");
		for (int j=0; j<nodestocompute[z][i]->c.lracard; j++){
			for (int k=0; k<nodestocompute[z][i]->c.lracompcard; k++){
				for (int l=0; l<6; l++){
					printf("%d, ", nodestocompute[z][i]->c.box[(j*nodestocompute[z][i]->c.lracompcard+k)*6+l]);
				}
				printf("   ");
			}
			printf("\n");
		}
		
	}
*/
	cutdata *c = (cutdata*)malloc(2*sizeof(cutdata));
	cutdata c1 = t->left->c;
	cutdata c2= t->right->c;
	int size=-1;	
	int amax=-1;
	int acmax=-1;
	int bmax=-1;
	int bcmax=-1;

	for (int i=0;i<c1.lracard;i++){
		for (int j=0;j<c2.lracard;j++){
			pointset p;
			p.size=0;
			p.members=(int*)malloc(c1.nrep*sizeof(int));
			for (int k=0; k<c1.nrep; k++){
				if (c1.lra[i*c1.nrep+k]==1){
					p.size++;
					p.members[p.size-1]=c1.tc[k];
				}
			} 
			pointset q;
			q.size=0;
			q.members=(int*)malloc(c2.nrep*sizeof(int));
			for (int k=0; k<c2.nrep; k++){
				if (c2.lra[i*c2.nrep+k]==1){
					q.size++;
					q.members[q.size-1]=c2.tc[k];
				}
			}
			int indexac=0;
			int indexbc=0;
			for (int k = 0; k<c2.nrep; k++){
				if (c2.lra[c2.nrep*j+k]==1){
					int rep = -1;
					for (int l=0; l< c1.nacomp; l++){
						if (c1.pointtorepincomp[2*l]==c2.tc[k]){
							rep=c1.pointtorepincomp[2*l+1];
							break;
						}
					}
					for (int l = 0; l< c1.nrepincomp; l++){
						if (c1.complementtc[l]==rep){
							rep=l;
							break;
						}
					}
				indexac = c1.mcomp[indexac*c1.nrepincomp+rep];
				}
			}
			for (int k = 0; k<c1.nrep; k++){
				if (c1.lra[c1.nrep*i+k]==1){
					int rep = -1;
					for (int l=0; l< c2.nacomp; l++){
						if (c2.pointtorepincomp[2*l]==c1.tc[k]){
							rep=c2.pointtorepincomp[2*l+1];
							break;
						}
					}
					for (int l = 0; l< c2.nrepincomp; l++){
						if (c2.complementtc[l]==rep){
							rep=l;
							break;
						}
					}
				indexbc = c2.mcomp[indexbc*c2.nrepincomp+rep];
				}
			}

			if ((size==-1)&&(c1.tab[i*c1.lracompcard+indexac]!=-1)&&(c2.tab[j*c2.lracompcard+indexbc]!=-1)){
				size=c1.tab[i*c1.lracompcard+indexac]+c2.tab[j*c2.lracompcard+indexbc];
				amax=i;
				acmax=indexac;
				bmax=j;
				bcmax=indexbc;
			}
			else {
				if ((c1.tab[i*c1.lracompcard+indexac]!=-1)&&(c2.tab[j*c2.lracompcard+indexbc]!=-1)){
					if (c1.tab[i*c1.lracompcard+indexac]+c2.tab[j*c2.lracompcard+indexbc]<size){
						size=c1.tab[i*c1.lracompcard+indexac]+c2.tab[j*c2.lracompcard+indexbc];
						amax=i;
						acmax=indexac;
						bmax=j;
						bcmax=indexbc;
					}
				}
			}
	
		}
		
	}
//	printf("Dominating set of size %d, amax=%d, acmax=%d, bmax=%d, bcmax=%d\n", size, amax, acmax, bmax, bcmax);

	
	int* left = (int*)malloc(c1.tab[amax*c1.lracompcard+acmax]*sizeof(int));
	int* right= (int*)malloc(c2.tab[bmax*c2.lracompcard+bcmax]*sizeof(int));
	left = computeDS (t->left, c1.tab[amax*c1.lracompcard+acmax], amax, acmax);
	right= computeDS (t->right, c2.tab[bmax*c2.lracompcard+bcmax], bmax, bcmax);

	for (int i = 0; i< c1.tab[amax*c1.lracompcard+acmax]; i++)
		set[i]=left[i];
	
	for (int i = 0; i< c2.tab[bmax*c2.lracompcard+bcmax]; i++)
		set[i+c1.tab[amax*c1.lracompcard+acmax]]=right[i];
	

/*	for (int i = 0; i<c2.tab[bmax*c2.lracompcard+bcmax]+c1.tab[amax*c1.lracompcard+acmax];i++)
		printf("(%d, %d)\n", g->pos[2*set[i]], g->pos[2*set[i]+1]);
	
	printf("\n");
*/
	return size;
}


int stepalgorithm (dectree* t, graph* g){
	
	if ((t->right==NULL)||(t->left==NULL)){
		t->c = cutThatTree (g, t);
		firstpreprocess (g,&(t->c));
		secondpreprocess (&(t->c), g);
		thirdpreprocess (&(t->c), g);

		t->c.tab = (int*)malloc(t->c.lracard*t->c.lracompcard*sizeof(int));
		t->c.box = (int*)malloc(6*t->c.lracard*t->c.lracompcard*sizeof(int));
		for (int i=0; i<6*t->c.lracard*t->c.lracompcard; i++)
			t->c.box[i]=-1;
		t->c.tab[0]=-1;
		t->c.tab[1]=0;
		t->c.tab[2]=1;
		t->c.tab[3]=1;
		t->computed=1;
	}

	else {
		if ((t->right->computed==1)&&(t->left->computed==1)){
			t->c = cutThatTree (g, t);
			firstpreprocess (g,&(t->c));
			secondpreprocess (&(t->c), g);
			thirdpreprocess (&(t->c), g);
			t->c.tab = (int*)malloc(t->c.lracard*t->c.lracompcard*sizeof(int));
			t->c.box = (int*)malloc(t->c.lracard*t->c.lracompcard*6*sizeof(int));
			for (int i = 0; i<t->c.lracard*t->c.lracompcard;i++){
				t->c.tab[i]=-1;
				t->c.box[6*i]=-1;
				t->c.box[6*i+1]=-1;
				t->c.box[6*i+2]=-1;
				t->c.box[6*i+3]=-1;
				t->c.box[6*i+4]=-1;
				t->c.box[6*i+5]=-1;
			}

			int *tmptab = (int*)malloc(5*t->right->c.lracard*t->left->c.lracard*t->c.lracompcard*sizeof(int));
			for (int i = 0; i< 5*t->right->c.lracard*t->left->c.lracard*t->c.lracompcard; i++)
				tmptab[i]=-1;

			int* tabg;
			int* lra;
			int* lrb;
			int* lrwc;
			int* lracard;
			int* lrbcard;
			int* lrwcard;
			int* lnracard;
			int* lnrbcard;
			int* lnrwcard;
			int* mw;
			int* macomp;
			int* mbcomp;
			int* nrepac;
			int* nrepbc;
			int* nrepwc;
			int* nrepa;
			int* nrepb;
			int* nrepw;
			int* repacomp;
			int* repbcomp;
			int* repwcomp;
			int* repa;
			int* repb;
			int* repw;
			int* taba;
			int* tabb;
			int* ptrac;
			int* ptrbc;
			int* ptrw;
			int* nacomp;
			int* nbcomp;
			int* nw;

			hipMalloc((void**)&tabg, 5*t->right->c.lracard*t->left->c.lracard*t->c.lracompcard*sizeof(int)); 
			hipMalloc((void**)&lra, t->left->c.lracard*t->left->c.nrep*sizeof(int)); 
			hipMalloc((void**)&lrb, t->right->c.lracard*t->right->c.nrep*sizeof(int));
			hipMalloc((void**)&lrwc, t->c.lracompcard*t->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&lracard, sizeof(int));
			hipMalloc((void**)&lrbcard, sizeof(int));
			hipMalloc((void**)&lrwcard, sizeof(int));
			hipMalloc((void**)&lnracard, sizeof(int));
			hipMalloc((void**)&lnrbcard, sizeof(int));
			hipMalloc((void**)&lnrwcard, sizeof(int));
			hipMalloc((void**)&mw, t->c.lracard*t->c.nrep*sizeof(int));
			hipMalloc((void**)&macomp, t->left->c.lracompcard*t->left->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&mbcomp, t->right->c.lracompcard*t->right->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&nrepac, sizeof(int));	
			hipMalloc((void**)&nrepbc, sizeof(int));
			hipMalloc((void**)&nrepwc, sizeof(int));
			hipMalloc((void**)&nrepa, sizeof(int));
			hipMalloc((void**)&nrepb, sizeof(int));
			hipMalloc((void**)&nrepw, sizeof(int));
			hipMalloc((void**)&repacomp, t->left->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&repbcomp, t->right->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&repwcomp, t->c.nrepincomp*sizeof(int));
			hipMalloc((void**)&repa, t->left->c.nrep*sizeof(int));
			hipMalloc((void**)&repb, t->right->c.nrep*sizeof(int));
			hipMalloc((void**)&repw, t->c.nrep*sizeof(int));
			hipMalloc((void**)&taba, t->left->c.lracard*t->left->c.lracompcard*sizeof(int));
			hipMalloc((void**)&tabb, t->right->c.lracard*t->right->c.lracompcard*sizeof(int));
			hipMalloc((void**)&ptrac, 2*t->left->c.nacomp*sizeof(int));
			hipMalloc((void**)&ptrbc, 2*t->right->c.nacomp*sizeof(int));
			hipMalloc((void**)&ptrw, 2*t->c.na*sizeof(int));
			hipMalloc((void**)&nacomp, sizeof(int));
			hipMalloc((void**)&nbcomp, sizeof(int));
			hipMalloc((void**)&nw, sizeof(int));

			hipMemcpy(tabg, tmptab, 5*t->right->c.lracard*t->left->c.lracard*t->c.lracompcard*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lra, t->left->c.lra, t->left->c.lracard*t->left->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lrb, t->right->c.lra, t->right->c.lracard*t->right->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lrwc, t->c.lracomp, t->c.lracompcard*t->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lracard, &(t->left->c.lracard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lrbcard, &(t->right->c.lracard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lrwcard, &(t->c.lracompcard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lnracard, &(t->left->c.lracompcard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lnrbcard, &(t->right->c.lracompcard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(lnrwcard, &(t->c.lnracompcard), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(mw, t->c.m, t->c.lracard*t->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(macomp, t->left->c.mcomp, t->left->c.lracompcard*t->left->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(mbcomp, t->right->c.mcomp, t->right->c.lracompcard*t->right->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepac, &(t->left->c.nrepincomp), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepbc, &(t->right->c.nrepincomp), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepwc, &(t->c.nrepincomp), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepa, &(t->left->c.nrep), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepb, &(t->right->c.nrep), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nrepw, &(t->c.nrep), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repacomp, t->left->c.complementtc, t->left->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repbcomp, t->right->c.complementtc, t->right->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repwcomp, t->c.complementtc, t->c.nrepincomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repa, t->left->c.tc, t->left->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repb, t->right->c.tc, t->right->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(repw, t->c.tc, t->c.nrep*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(taba, t->left->c.tab, t->left->c.lracard*t->left->c.lracompcard*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(tabb, t->right->c.tab, t->right->c.lracard*t->right->c.lracompcard*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(ptrac, t->left->c.pointtorepincomp, 2*t->left->c.nacomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(ptrbc, t->right->c.pointtorepincomp, 2*t->right->c.nacomp*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(ptrw, t->c.pointtorep, 2*t->c.na*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nacomp, &(t->left->c.nacomp), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nbcomp, &(t->right->c.nacomp), sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(nw, &(t->c.na), sizeof(int), hipMemcpyHostToDevice);

			computeAlgorithm <<<t->c.lracompcard, t->left->c.lracard*t->right->c.lracard>>> (tabg, lra, lrb, lrwc, lracard, lrbcard, lrwcard, lnracard, lnrbcard, lnrwcard, mw, macomp, mbcomp, nrepac, nrepbc, nrepwc, nrepa, nrepb, nrepw, repacomp, repbcomp,repwcomp, repa, repb, repw, taba, tabb, ptrac, ptrbc, ptrw, nacomp, nbcomp, nw);		

			hipMemcpy(tmptab, tabg, 5*t->right->c.lracard*t->left->c.lracard*t->c.lracompcard*sizeof(int), hipMemcpyDeviceToHost);			

			for (int i =0; i<t->c.lracompcard; i++){
				for (int j = 0; j<t->left->c.lracard; j++){
					for (int k = 0; k<t->right->c.lracard; k++){
						if ((t->c.tab[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*t->c.lracompcard+i]==-1)&&(tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3]!=-1)&&(tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4]!=-1)){
							t->c.tab[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*t->c.lracompcard+i]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3]+tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4];
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i]=j;
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+1]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+1];
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+2]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3];
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+3]=k;
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+4]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+2];
							t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+5]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4];
						}
						else {
							if ((tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3]!=-1)&&(tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4]!=-1)&&(tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3]+tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4]<t->c.tab[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*t->c.lracompcard+i])){
								t->c.tab[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*t->c.lracompcard+i]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3]+tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4];
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i]=j;
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+1]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+1];
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+2]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+3];
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+3]=k;
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+4]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+2];
								t->c.box[tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5]*6*t->c.lracompcard+6*i+5]=tmptab[5*t->left->c.lracard*t->right->c.lracard*i+j*5*t->right->c.lracard+k*5+4];
							}
						}
					}
				}
			}
			t->computed=1;
		}	
		else {
			t->computed=0;
		
		}	
	}

	return EXIT_SUCCESS;
}

int* computeDS (dectree* t, int much, int aleft, int acleft){
	int* sol = (int*)malloc(much*sizeof(int));

	if ((t->left==NULL)&&(t->right==NULL)&&(much==1)){
		sol[0]=t->c.tc[0];
	}

	else if (much!=0){
		int *left=(int*)malloc(t->c.box[6*t->c.lracompcard*aleft+6*acleft+2]);
		left = computeDS (t->left, t->c.box[6*t->c.lracompcard*aleft+6*acleft+2], t->c.box[6*t->c.lracompcard*aleft+6*acleft+0], t->c.box[6*t->c.lracompcard*aleft+6*acleft+1]);
	

		int *right=(int*)malloc(t->c.box[6*t->c.lracompcard*aleft+6*acleft+5]);
		right = computeDS (t->right, t->c.box[6*t->c.lracompcard*aleft+6*acleft+5], t->c.box[6*t->c.lracompcard*aleft+6*acleft+3], t->c.box[6*t->c.lracompcard*aleft+6*acleft+4]);
		for (int i = 0; i<t->c.box[6*t->c.lracompcard*aleft+6*acleft+2]; i++){
			sol[i]=left[i];
		}
		for (int i = 0; i<t->c.box[6*t->c.lracompcard*aleft+6*acleft+5]; i++){
			sol[t->c.box[6*t->c.lracompcard*aleft+6*acleft+2]+i]=right[i];
		}
	
	}

	
	return sol;
}

int getBW (dectree* t, graph* g, int z){
	int bwmax=-1;
	if ((t->right==NULL)||(t->left==NULL))
		bwmax=2;
	else {
		int n = getnumberofnodes (t) -1;
		tocompute[z]=0;
		int* sizes= (int*)malloc(n*sizeof(int));
		nodestocompute[z]=(dectree**)malloc(n*sizeof(dectree*));
		fillThevoid(t->right, g, z);
		fillThevoid(t->left, g, z);

		for (int i=0; i<n; i++){
			nodestocompute[z][i]->c = cutThatTree (g, nodestocompute[z][i]);

			firstpreprocess(g, &(nodestocompute[z][i]->c));
			secondpreprocess(&(nodestocompute[z][i]->c), g);
			sizes[i]=nodestocompute[z][i]->c.lracard;

		}
		bwmax=sizes[0];
		for (int i=1; i<n; i++){
			if (sizes[i]>bwmax)
				bwmax = sizes[i];
		}
	}
	return bwmax;
}
